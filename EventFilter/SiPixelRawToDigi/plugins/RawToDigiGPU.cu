#include "hip/hip_runtime.h"
/**2017-03-02  Sushil Dubey  <sdubey@felk40.cern.ch>
 *
 * File Name: RawToDigiGPU.cu
 * Description: It converts Raw data into Digi data using GPU
 * then it applies the adc threshold to drop the dead pixels
 * The Output of RawToDigi data is given to pixelClusterizer
 *
**/ 
// System includes
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <assert.h>
#include <iomanip>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "CudaError.h"
#include "EventInfoGPU.h"
#include "RawToDigiGPU.h"
#include "RawToDigiMem.h"
using namespace std;

// forward declaration to be moved in header file
//void PixelCluster_Wrapper(uint *xx_adc, uint *yy_adc, uint *adc_d,const uint wordCounter, 
//                          const int *mIndexStart, const int *mIndexEnd);

/*
  This functions checks for cuda error
  Input: debug message
  Output: returns cuda error message
*/
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(-1);
  }
}

/*
void initCablingMap() {

  ifstream mapFile;
  mapFile.open("RawId_ModuleId_CablingMap_ArrayFile.txt");
  string str;
  getline(mapFile, str);
  uint rawId, moduleId, rocInDU;
  int i =1;  // cabling map index starts at 1
  while(!mapFile.eof()) {
    mapFile >> rawId >> rocInDU >> moduleId;
    Map->RawId[i]    = rawId;
    Map->rocInDet[i] = rocInDU;
    Map->moduleId[i] = moduleId;
    i++;
  }
  mapFile.close();
}
*/
// New cabling Map
void initCablingMap() {

  ifstream mapFile;
  mapFile.open("Pixel_Phase1_Raw2Digi_GPU_Cabling_Map.dat");
  string str;
  getline(mapFile, str);
  uint Index, FedId, Link, idinLNK, B_F, RawID, idinDU, ModuleID;
  int i =1;  // cabling map index starts at 1
  while(!mapFile.eof()) {
    mapFile >> Index>>FedId>>Link>>idinLNK>>B_F>>RawID>>idinDU>>ModuleID;
    Map->RawId[i] = RawID;
    Map->rocInDet[i] = idinDU;
    Map->moduleId[i] = ModuleID;
    i++;
  }
  mapFile.close();
  cout<<"Cabling Map uploaded successfully!"<<endl;
}

void initDeviceMemory() {
  int sizeByte = MAX_FED * MAX_LINK * MAX_ROC * sizeof(uint)+sizeof(uint);
  // Unified memory for cabling map
  hipMallocManaged((void**)&Map,  sizeof(CablingMap));
  hipMallocManaged((void**)&Map->RawId,    sizeByte);
  hipMallocManaged((void**)&Map->rocInDet, sizeByte);
  hipMallocManaged((void**)&Map->moduleId, sizeByte);
    // Number of words for all the feds 
  uint MAX_WORD_SIZE = MAX_FED*MAX_WORD*NEVENT*sizeof(uint); 
  uint FSIZE = 2*MAX_FED*NEVENT*sizeof(uint)+sizeof(uint);
  

  int MSIZE = NMODULE*NEVENT*sizeof(int);

  hipMalloc((void**)&eventIndex_d, (NEVENT+1)*sizeof(uint));

  hipMalloc((void**)&word_d,       MAX_WORD_SIZE);
  hipMalloc((void**)&fedIndex_d,   FSIZE);
  hipMalloc((void**)&xx_d,         MAX_WORD_SIZE); // to store the x and y coordinate
  hipMalloc((void**)&yy_d,         MAX_WORD_SIZE);
  hipMalloc((void**)&xx_adc,       MAX_WORD_SIZE); // to store the x and y coordinate
  hipMalloc((void**)&yy_adc,       MAX_WORD_SIZE);
  hipMalloc((void**)&adc_d,        MAX_WORD_SIZE);
  hipMalloc((void**)&layer_d ,     MAX_WORD_SIZE);

  hipMalloc((void**)&moduleId_d,   MAX_WORD_SIZE);
  hipMalloc((void**)&mIndexStart_d, MSIZE);
  hipMalloc((void**)&mIndexEnd_d,   MSIZE);
  
  cout<<"Memory Allocated successfully !\n";
  // Upload the cabling Map
  initCablingMap();
  
}

void freeMemory() {

  //GPU specific
  hipFree(eventIndex_d);
  hipFree(word_d);
  hipFree(fedIndex_d);
  hipFree(adc_d);
  hipFree(layer_d);
  hipFree(xx_d);
  hipFree(yy_d);
  hipFree(xx_adc);
  hipFree(yy_adc);
  
  hipFree(moduleId_d);
  hipFree(mIndexStart_d);
  hipFree(mIndexEnd_d);

  hipFree(Map->RawId);
  hipFree(Map->rocInDet); 
  hipFree(Map->moduleId);
  hipFree(Map);
  cout<<"Memory Released !\n";

}

__device__ uint getLink(uint ww)  {
  //printf("Link_shift: %d  LINK_mask: %d\n", LINK_shift, LINK_mask);
  return ((ww >> LINK_shift) & LINK_mask);
}

__device__ uint getRoc(uint ww) {
  return ((ww >> ROC_shift ) & ROC_mask);
}
__device__ uint getADC(uint ww) {
  return ((ww >> ADC_shift) & ADC_mask);
}

__device__ bool isBarrel(uint rawId) {
  return (1==((rawId>>25)&0x7));
}
//__device__ uint FED_START = 1200;

__device__ DetIdGPU getRawId(const CablingMap *Map, uint fed, uint link, uint roc) {
  uint index = fed * MAX_LINK* MAX_ROC + (link-1)* MAX_ROC + roc;
  DetIdGPU detId = {Map->RawId[index], Map->rocInDet[index], Map->moduleId[index]};
  return detId;  
}

//reference http://cmsdoxygen.web.cern.ch/cmsdoxygen/CMSSW_9_2_0/doc/html/dd/d31/FrameConversion_8cc_source.html
//http://cmslxr.fnal.gov/source/CondFormats/SiPixelObjects/src/PixelROC.cc?v=CMSSW_9_2_0#0071
// Convert local pixel to global pixel
__device__ Pixel frameConversion(bool bpix, int side, uint layer,uint rocIdInDetUnit, Pixel local) {
  
  int slopeRow  = 0,  slopeCol = 0;
  int rowOffset = 0, colOffset = 0;

  if(bpix) {
    
    if(side==-1 && layer!=1) { // -Z side: 4 non-flipped modules oriented like 'dddd', except Layer 1
      if (rocIdInDetUnit <8) {
        slopeRow = 1;     
        slopeCol = -1;
        rowOffset = 0;
        colOffset = (8-rocIdInDetUnit)*numColsInRoc-1;
      }
      else {
        slopeRow  = -1;
        slopeCol  = 1;
        rowOffset = 2*numRowsInRoc-1;
        colOffset = (rocIdInDetUnit-8)*numColsInRoc;
      } // if roc
    }
    else { // +Z side: 4 non-flipped modules oriented like 'pppp', but all 8 in layer1
      if(rocIdInDetUnit <8) {
        slopeRow  = -1;
        slopeCol  =  1;
        rowOffset = 2*numRowsInRoc-1;
        colOffset = rocIdInDetUnit * numColsInRoc;
      }
      else {
        slopeRow  = 1;
        slopeCol  = -1;
        rowOffset = 0;
        colOffset = (16-rocIdInDetUnit)*numColsInRoc-1;
      }
    }

  }
  else { // fpix
    if(side==-1) { // pannel 1
      if (rocIdInDetUnit < 8) {
        slopeRow = 1;
        slopeCol = -1;
        rowOffset = 0;
        colOffset = (8-rocIdInDetUnit)*numColsInRoc-1;
      }
      else {
        slopeRow = -1;
        slopeCol = 1;
        rowOffset = 2*numRowsInRoc-1;
        colOffset = (rocIdInDetUnit-8)*numColsInRoc;
      }
    }
    else { // pannel 2
      if (rocIdInDetUnit < 8) {
        slopeRow = 1;
        slopeCol = -1;
        rowOffset = 0;
        colOffset = (8-rocIdInDetUnit)*numColsInRoc-1;
      }
      else {
        slopeRow = -1;
        slopeCol = 1;
        rowOffset = 2*numRowsInRoc-1;
        colOffset = (rocIdInDetUnit-8)*numColsInRoc;
      }

    } // side

  }

  uint gRow = rowOffset+slopeRow*local.row;
  uint gCol = colOffset+slopeCol*local.col;
  //printf("Inside frameConversion gRow: %u  gCol: %u\n",gRow, gCol);
  Pixel global = {gRow, gCol};
  return global;
}


/*----------
* Name: applyADCthreshold_kernel()
* Desc: converts adc count to electrons and then applies the 
* threshold on each channel. 
* make pixel to 0 if it is below the threshold
* Input: xx_d[], yy_d[], layer_d[], wordCounter, adc[], ADCThreshold
*-----------
* Output: xx_adc[], yy_adc[] with pixel threshold applied 
*/
// kernel to apply adc threshold on the channels  
__global__ void applyADCthreshold_kernel
(const uint *xx_d, const uint *yy_d, const uint *layer_d, uint *adc, const uint wordCounter,
 const ADCThreshold adcThreshold, uint *xx_adc, uint *yy_adc ) {
  int tid = threadIdx.x;
  int gIndex = blockDim.x*blockIdx.x+tid;
  if(gIndex<wordCounter) {
    //int i=0;
    //for(DigiIterator di = begin; di != end; ++di) {
      uint adcOld = adc[gIndex];
      const float gain = adcThreshold.theElectronPerADCGain_; // default: 1 ADC = 135 electrons
      const float pedestal = 0; //
      int adcNew = int(adcOld*gain+pedestal);
      // rare chance of entering into the if()
      if (layer_d[gIndex]>=adcThreshold.theFirstStack_) {
        if (adcThreshold.theStackADC_==1 && adcOld==1) {
          adcNew = int(255*135); // Arbitrarily use overflow value.
        }
        if (adcThreshold.theStackADC_ >1 && adcThreshold.theStackADC_!=255 && adcOld>=1){
          adcNew = int((adcOld-1) * gain * 255/float(adcThreshold.theStackADC_-1));
        }
      }
  
    if(adcNew >adcThreshold.thePixelThreshold ) {
      xx_adc[gIndex]=xx_d[gIndex];
      yy_adc[gIndex]=yy_d[gIndex];
    }
    else {
      xx_adc[gIndex]=0; // 0: dead pixel
      yy_adc[gIndex]=0;
    }
    adc[gIndex] = adcNew;
  }
}  


// Kernel to perform Raw to Digi conversion
__global__ void RawToDigi_kernel(const CablingMap *Map,const uint *Word,const uint *fedIndex, 
                                 uint *eventIndex,const uint stream, uint *XX, uint *YY, uint *moduleId, int *mIndexStart, 
                                 int *mIndexEnd, uint *ADC, uint *layerArr, uint *RawId) 
{
  uint blockId  = blockIdx.x;
  uint eventno  = blockIdx.y + gridDim.y*stream;
  
  //const uint eventOffset  = eventIndex[eventno]; 
  uint fedOffset    = 2*150*eventno;

  uint fedId     = fedIndex[fedOffset+blockId];
  uint threadId  = threadIdx.x;
  
  uint begin  = fedIndex[fedOffset + MAX_FED+blockId];
  uint end    = fedIndex[fedOffset + MAX_FED+blockId+1];

  if(blockIdx.x==gridDim.x-1) {
    end = eventIndex[eventno+1]; // for last fed to get the end index
  }

  //if(threadId==0) printf("Event: %u blockId: %u start: %u end: %u\n", eventno, blockId, begin, end);
  int no_itr = (end - begin)/blockDim.x + 1; // to deal with number of hits greater than blockDim.x 
  #pragma unroll
  for(int i =0; i<no_itr; i++) { // use a static number to optimize this loop
    uint gIndex = begin + threadId + i*blockDim.x; 
    if(gIndex <end) {
      uint ww    = Word[gIndex]; // Array containing 32 bit raw data
      if(ww == 0 ) {
        //noise and dead channels are ignored
        XX[gIndex]    = 0;  // 0 is an indicator of a noise/dead channel
        YY[gIndex]    = 0; // skip these pixels during clusterization
        ADC[gIndex]   = 0;
        layerArr[gIndex] = fedId;//0; 
        moduleId[gIndex] = 9999; //9999 is the indication of bad module, taken care later
        RawId[gIndex] = 0;  
        continue ;         // 0: bad word, 
      } 
      uint link  = getLink(ww);            // Extract link
      uint roc   = getRoc(ww);             // Extract Roc in link
      DetIdGPU detId = getRawId(Map, fedId, link, roc);
      uint rawId  = detId.RawId;
      uint rocIdInDetUnit = detId.rocInDet;
     
      bool barrel = isBarrel(rawId);
  
      uint layer =0;//, ladder =0;
      int side =0, panel =0, module=0;//disk =0,blade =0
    
      if(barrel) {
        layer  = (rawId >> layerStartBit_)  & layerMask_;
        //ladder = (rawId >> ladderStartBit_) & ladderMask_;
        module = (rawId >> moduleStartBit_) & moduleMask_;
        side   = (module<5)? -1:1;
     
      }
      else {
        // endcap ids
        layer = 0;
        panel = (rawId >> panelStartBit_) & panelMask_;
        //disk  = (rawId >> diskStartBit_)  & diskMask_ ;
        side  = (panel==1)? -1:1;
        //blade = (rawId>>bladeStartBit_) & bladeMask_;
      }
      // ***special case of layer to 1 be handled here
      Pixel localPix;
      if(layer==1) {
        uint col = (ww >> COL_shift) & COL_mask;
        uint row = (ww >> ROW_shift) & ROW_mask;
        localPix.row = row;
        localPix.col = col;
        //if(event==0 && fedId==0)
         //printf("col: %u  row: %u\n",col, row);
      }
      else {
        // ***conversion rules for dcol and pxid
        uint dcol = (ww >> DCOL_shift) & DCOL_mask;
        uint pxid = (ww >> PXID_shift) & PXID_mask;
        uint row  = numRowsInRoc - pxid/2;
        uint col  = dcol*2 + pxid%2;
        localPix.row = row;
        localPix.col = col;
      }
      //if(fedId==48)
        //printf("%14u%6d%6d%6d\n",ww,localPix.row,localPix.col, getADC(ww));
      
      Pixel globalPix = frameConversion(barrel, side, layer,rocIdInDetUnit, localPix);
      //if(threadId==0) printf("Event: %u fedId: %u\n",eventno, fedId );
      XX[gIndex]    = globalPix.row;// +1  ; // origin shifting by 1 0-159
      YY[gIndex]    = globalPix.col;// +1 ; // origin shifting by 1 0-415
      ADC[gIndex]   = getADC(ww);
      layerArr[gIndex] = fedId;//layer;
      moduleId[gIndex] = detId.moduleId;
      RawId[gIndex] = detId.RawId;
    } // end of if(gIndex < end)
  } // end of for(int i =0;i<no_itr...)
  
  __syncthreads();

  // three cases possible
  // case 1: 21 21 21 22 21 22 22
  // pos   : 0  1  2  3  4  5  6
  // solution swap 21 with 22 : 21 21 21 21 22 22 22
  // atomicExch(address, value), set the variable at address to value.
  // do the swapping for above case and replace the 9999 with 
  // valid moduleId
   
  for(int i =0; i<no_itr; i++) { 
    uint gIndex = begin + threadId + i*blockDim.x;  
    if(gIndex <end) {
      //rare condition 
      if(moduleId[gIndex]==moduleId[gIndex+2] && moduleId[gIndex]<moduleId[gIndex+1]) {
        atomicExch(&moduleId[gIndex+2], atomicExch(&moduleId[gIndex+1], moduleId[gIndex+2]));
        //*swap all the digi id
        atomicExch(&XX[gIndex+2], atomicExch(&XX[gIndex+1], XX[gIndex+2]));
        atomicExch(&YY[gIndex+2], atomicExch(&YY[gIndex+1], YY[gIndex+2]));
        atomicExch(&ADC[gIndex+2], atomicExch(&ADC[gIndex+1], ADC[gIndex+2]));
        atomicExch(&RawId[gIndex+2], atomicExch(&RawId[gIndex+1], RawId[gIndex+2]));
        atomicExch(&layerArr[gIndex+2], atomicExch(&layerArr[gIndex+1], layerArr[gIndex+2]));
         
      }
      __syncthreads();
      //rarest condition
      // above condition fails at 361 361 361 363 362 363 363
      // here we need to swap 362 with previous 363
      if(moduleId[gIndex]==moduleId[gIndex+2] && moduleId[gIndex]>moduleId[gIndex+1]) {
        atomicExch(&moduleId[gIndex+1], atomicExch(&moduleId[gIndex], moduleId[gIndex+1]));
        //*swap all the digi id
        atomicExch(&XX[gIndex+1], atomicExch(&XX[gIndex], XX[gIndex+1]));
        atomicExch(&YY[gIndex+1], atomicExch(&YY[gIndex], YY[gIndex+1]));
        atomicExch(&ADC[gIndex+1], atomicExch(&ADC[gIndex], ADC[gIndex+1]));
        atomicExch(&RawId[gIndex+1], atomicExch(&RawId[gIndex], RawId[gIndex+1]));
        atomicExch(&layerArr[gIndex+1], atomicExch(&layerArr[gIndex], layerArr[gIndex+1]));
         
      }
      // moduleId== 9999 then pixel is bad with x=y=layer=adc=0
      // this bad pixel will not affect the cluster, since for cluster
      // the origin is shifted at (1,1) so x=y=0 will be ignored
      // assign the previous valid moduleId to this pixel to remove 9999
      // so that we can get the start & end index of module easily.
      __syncthreads(); // let the swapping finish first
      if(moduleId[gIndex]==9999) {
        int m=gIndex;
        while(moduleId[--m]==9999) {} //skip till you get the valid module
        moduleId[gIndex]=moduleId[m];
      } 
    } // end of if(gIndex<end)
  } //  end of for(int i=0;i<no_itr;...)
  __syncthreads();

  // mIndexStart stores staring index of module 
  // mIndexEnd stores end index of module 
  // both indexes are inclusive 
  // check consecutive module numbers
  // for start of fed
  for(int i =0; i<no_itr; i++) { 
    uint gIndex = begin + threadId + i*blockDim.x; 
    uint moduleOffset = NMODULE*eventno; 
    //if(threadId==0) printf("moduleOffset: %u\n",moduleOffset );
    if(gIndex <end) {
      if(gIndex == begin) {
        mIndexStart[moduleOffset+moduleId[gIndex]] = gIndex;
      }
      // for end of the fed
      if(gIndex == (end-1)) {  
        mIndexEnd[moduleOffset+moduleId[gIndex]] = gIndex;
      }   
      // point to the gIndex where two consecutive moduleId varies
      if(gIndex!= begin && (gIndex<(end-1)) && moduleId[gIndex]!=9999) {
        if(moduleId[gIndex]<moduleId[gIndex+1] ) {
          mIndexEnd[moduleOffset + moduleId[gIndex]] = gIndex;
        }
        if(moduleId[gIndex] > moduleId[gIndex-1] ) {
          mIndexStart[moduleOffset+ moduleId[gIndex]] = gIndex;
        } 
      } //end of if(gIndex!= begin && (gIndex<(end-1)) ...  
    } //end of if(gIndex <end) 
  }
} // end of Raw to Digi kernel

// kernel wrapper called from runRawToDigi_kernel
void RawToDigi_Cluster_CPE_wrapper (const uint wordCounter, uint *word, 
                                    const uint fedCounter,  uint *fedIndex,
                                    uint *eventIndex) { 
  
 
  cout<<"Inside RawToDigi , total words: "<<wordCounter<<endl;
  const int NSTREAM = 4;
  hipStream_t stream[NSTREAM];
  for(int i=0;i<NSTREAM;i++) {
    hipStreamCreate(&stream[i]);
  }

  //const int nBlocks = fedCounter; // =108
  const int threads = 512;
  const int blockX = 108; // 108 feds
  const int blockY = 2;   //blockIdx.y=2 is the no of events processed in kernel concurrently
  dim3 gridsize(blockX, blockY); 
  //fedIndex[MAX_FED+nBlocks] = wordCounter;
  
  int MSIZE = NMODULE*NEVENT*sizeof(int);
  // initialize moduleStart & moduleEnd with some constant(-1)
  // just to check if it updated in kernel or not
  hipMemset(mIndexStart_d, -1, MSIZE);
  hipMemset(mIndexEnd_d, -1, MSIZE);
  hipMemcpy(eventIndex_d, eventIndex, (NEVENT+1)*sizeof(uint), hipMemcpyHostToDevice);

  // for debugging purpose only
  uint *RawId_d;
  hipMalloc((void**)&RawId_d, wordCounter*sizeof(uint));
  
  //for(int i=0;i<NEVENT;i++) cout<<"Event: "<<i<<" offset: "<<
  int FSIZE = (blockY*2*MAX_FED +1)*sizeof(uint); // 0 to 150:fedId, 150:300: fedIndex
  
  int fedOffset  = 0;
  int wordOffset = 0;
  int wordSize   = 0;
  for (int i=0; i<NSTREAM; i++) {
    fedOffset  = blockY*2*150*i;
    wordOffset = eventIndex[blockY*i];
    // total no of words in blockY event to be trasfered on device 
    wordSize   = (eventIndex[blockY*(i+1)] - eventIndex[blockY*i]); 

    hipMemcpyAsync(&word_d[wordOffset], &word[wordOffset], wordSize*sizeof(uint), hipMemcpyHostToDevice, stream[i]);

    hipMemcpyAsync(&fedIndex_d[fedOffset], &fedIndex[fedOffset], FSIZE, hipMemcpyHostToDevice, stream[i]); 
    // Launch rawToDigi kernel
    RawToDigi_kernel<<<gridsize,threads,0, stream[i]>>>(Map,word_d, fedIndex_d,eventIndex_d,i, xx_d, yy_d, moduleId_d,
                                        mIndexStart_d, mIndexEnd_d, adc_d,layer_d, RawId_d);
  }
  //hipDeviceSynchronize();
  checkCUDAError("Error in RawToDigi_kernel");
  for (int i = 0; i<NSTREAM; i++) {
    hipStreamSynchronize(stream[i]);
  }

  // some extra stuff is done in the kernel for output purpose
  uint size = wordCounter*sizeof(uint);
  uint *xx,*yy,*adc,*fedId,*RawId;
  xx = (uint*)malloc(wordCounter*sizeof(uint));
  yy = (uint*)malloc(wordCounter*sizeof(uint));
  adc = (uint*)malloc(wordCounter*sizeof(uint));
  fedId = (uint*)malloc(wordCounter*sizeof(uint));
  RawId = (uint*)malloc(wordCounter*sizeof(uint));
  hipMemcpy(xx,xx_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(yy,yy_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(adc,adc_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(fedId,layer_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(RawId,RawId_d, size, hipMemcpyDeviceToHost);
  //ofstream r2d("R2D_debug_fedId_xyadc_moduleId.txt");
  ofstream R2D("R2D_GPU.txt");
  R2D<<setw(6)<<"fedId"<<setw(14)<<"RawId"<<setw(6)<<"xx"<<setw(6)<<"yy"<<setw(6)<<"adc"<<endl;
  for(int i=0;i<wordCounter;i++) {
    R2D<<setw(6)<<fedId[i]+1200<<setw(14)<<RawId[i]<<setw(6)<<xx[i]<<setw(6)<<yy[i]<<setw(6)<<adc[i]<<endl;
  }
  R2D.close();
  free(xx);
  free(yy);
  free(adc);
  free(fedId);
  free(RawId); 
  hipFree(RawId_d);
  
  /*
  checkCUDAError("Error in memcpy for moduleStart_end H2D");
  // kernel to apply adc threashold on the channel
  ADCThreshold adcThreshold;
  uint numThreads = 512;
  uint numBlocks = wordCounter/512 +1;
  applyADCthreshold_kernel<<<numBlocks, numThreads>>>(xx_d, yy_d,layer_d,adc_d,wordCounter,adcThreshold, xx_adc, yy_adc);
  hipDeviceSynchronize();
  checkCUDAError("Error in applying ADC threshold");
  */
  //PixelCluster_Wrapper(xx_adc , yy_adc, adc_d,wordCounter, mIndexStart_d, mIndexEnd_d);
}
